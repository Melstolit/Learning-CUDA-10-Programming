// Demonstration of basic CUDA error handling.
// Example fgor video 2.5.


#include <hip/hip_runtime.h>
#include <stdio.h>

// Standard CUDA API functions
#include <hip/hip_runtime_api.h>

// Error checking macro
#define cudaCheckError(code)                                             \
  {                                                                      \
    if ((code) != hipSuccess) {                                         \
      fprintf(stderr, "Cuda failure %s:%d: '%s' \n", __FILE__, __LINE__, \
              hipGetErrorString(code));                                 \
    }                                                                    \
  }

__global__ void bad()
{
  char *x = nullptr;
  *x = 1;
}

__global__ void good() {}

int main()
{
  int *foo = nullptr;
  size_t size = 1lu << 33;
  hipError_t status = hipMalloc(&foo, size);
  const char *message = hipGetErrorString(status);

  status = hipGetLastError();

  status = hipMalloc(&foo, 16);
  message = hipGetErrorString(status);

  bad<<<1, 1>>>();
  status = hipDeviceSynchronize();
  message = hipGetErrorString(status);

  good<<<1, 16>>>();
  status = hipDeviceSynchronize();
  message = hipGetErrorString(status);

  cudaCheckError(hipMalloc(&foo, 16))

      return 0;
}
