#include "hip/hip_runtime.h"
// Demonstration of the unified virtual address space. Run multiple scans
// concurrently across all available devices
// Example for video 6.5.

#include <assert.h>
#include <iostream>
#include <memory>
#include <numeric>
#include <random>

// Standard CUDA API functions
#include <hip/hip_runtime_api.h>

// CUDA cooperative groups API
#include <hip/hip_cooperative_groups.h>

#include "../utils.h"

void scan_reference(const int *source, int *dest, unsigned int count)
{
  int sum = 0;
  for (int i = 0; i < count; i++) {
    sum += source[i];
    dest[i] = sum;
  }
}

const int BLOCK_SIZE = 1024;

// Scan using shared memory, within a single block.
__device__ int block_scan(int idata, int shared_data[],
                          cooperative_groups::thread_block block)
{
  // Index into shared memory
  int si = threadIdx.x;
  shared_data[si] = 0;
  si += blockDim.x;
  shared_data[si] = idata;

  for (int offset = 1; offset < blockDim.x; offset *= 2) {
    cooperative_groups::sync(block);
    int t = shared_data[si] + shared_data[si - offset];
    cooperative_groups::sync(block);
    shared_data[si] = t;
  }

  return shared_data[si];
}

// First step of scan: process each block separately
__global__ void scan1(const int *source, int *dest)
{
  // Shared memory buffer. By allocating extra elements we avoid bounds
  // checks on shared memory access.
  __shared__ int shared_data[2 * BLOCK_SIZE];

  // Index into global memory
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  // Load data from global memory
  int idata = source[index];

  // Shared memory scan within this block
  int result =
      block_scan(idata, shared_data, cooperative_groups::this_thread_block());

  // Write back to global memory
  dest[index] = result;
}

// Second step of scan: compute prefix sums for each block
__global__ void scan2(const int *dest, int *block_sums, unsigned int count)
{
  // Shared memory buffer. By allocating extra elements we avoid bounds
  // checks on shared memory access.
  __shared__ int shared_data[2 * BLOCK_SIZE];

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  int idata = (index == 0) ? 0 : dest[index * blockDim.x - 1];
  block_sums[index] =
      block_scan(idata, shared_data, cooperative_groups::this_thread_block());
}

// Final step of scan: add block sums to every result.
__global__ void finish_scan(const int *block_sums, int *dest)
{
  __shared__ int block_sum;

  if (threadIdx.x == 0) {
    block_sum = block_sums[blockIdx.x];
  }
  cooperative_groups::sync(cooperative_groups::this_thread_block());

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  dest[index] += block_sum;
}

static void print_pointer(const std::string &name, const void *pointer)
{
  hipPointerAttribute_t attributes;
  auto result = hipPointerGetAttributes(&attributes, pointer);

  std::cout << name << ": ";
  if (result != hipSuccess) {
    std::cout << "get attributes failed";
    return;
  } else {
    switch (attributes.type) {
      case cudaMemoryTypeUnregistered:
        std::cout << "unregistered";
        break;
      case hipMemoryTypeHost:
        std::cout << "host memory";
        break;
      case hipMemoryTypeDevice:
        std::cout << "device " << attributes.device;
        break;
      case hipMemoryTypeManaged:
        std::cout << "managed";
        break;
    }
  }

  std::cout << "\n";
}

int main(int argc, char **argv)
{
  // Maximum possible size with two-level scan.
  const unsigned int COUNT = BLOCK_SIZE * BLOCK_SIZE;
  const int N_STREAMS = 2;

  int *sources[N_STREAMS], *dests[N_STREAMS];

  // Fill source arrays with some arbitrary test values
  std::mt19937 rng;
  rng.seed(0);
  std::uniform_int_distribution<std::mt19937::result_type> dist(0, 9);

  int device_count;
  cudaCheckError(hipGetDeviceCount(&device_count));

  for (int i = 0; i < N_STREAMS; i++) {
    // Allocate page-locked memory to allow asynchronous transfers.
    hipHostMalloc(&sources[i], COUNT * sizeof(int));
    hipHostMalloc(&dests[i], COUNT * sizeof(int));
    for (int j = 0; j < COUNT; j++) {
      sources[i][j] = dist(rng);
    }
  }

  // Allocate device memory and transfer data
  int n_blocks1 = (COUNT + BLOCK_SIZE - 1) / BLOCK_SIZE;

  int *sources_dev[N_STREAMS], *dests_dev[N_STREAMS], *block_sums[N_STREAMS];
  size_t size = COUNT * sizeof(int);
  hipStream_t stream[N_STREAMS];

  for (int i = 0; i < N_STREAMS; i++) {
    int device = i % device_count;
    cudaCheckError(hipSetDevice(device));
    cudaCheckError(hipStreamCreate(&stream[i]));
    cudaCheckError(hipMalloc(&sources_dev[i], size));
    cudaCheckError(hipMalloc(&dests_dev[i], size));
    // Temporary buffer for kernels
    cudaCheckError(hipMalloc(&block_sums[i], n_blocks1 * sizeof(int)));
  }

  {
    KernelTimer t;

    for (int i = 0; i < N_STREAMS; i++) {
      int device = i % device_count;
      cudaCheckError(hipSetDevice(device));

      std::cout << "Stream " << i << " on device " << device << "\n";
      print_pointer("source", sources[i]);
      print_pointer("source_dev", sources_dev[i]);
      print_pointer("dest_dev", dests_dev[i]);
      print_pointer("dest", dests[i]);

      // Copy data to device
      cudaCheckError(hipMemcpyAsync(sources_dev[i], sources[i], size,
                                     hipMemcpyDefault, stream[i]));

      // Run the scan
      scan1<<<n_blocks1, BLOCK_SIZE, 0, stream[i]>>>(sources_dev[i],
                                                     dests_dev[i]);

      int n_blocks2 = (n_blocks1 + BLOCK_SIZE - 1) / BLOCK_SIZE;
      assert(n_blocks2 == 1);
      scan2<<<n_blocks2, BLOCK_SIZE, 0, stream[i]>>>(dests_dev[i],
                                                     block_sums[i], n_blocks1);

      finish_scan<<<n_blocks1, BLOCK_SIZE, 0, stream[i]>>>(block_sums[i],
                                                           dests_dev[i]);

      // Copy results back to the host
      cudaCheckError(hipMemcpyAsync(dests[i], dests_dev[i], size,
                                     hipMemcpyDefault, stream[i]));
      std::cout << "\n";
    }
  }

  for (int i = 0; i < N_STREAMS; i++) {
    cudaCheckError(hipFree(sources_dev[i]));
    cudaCheckError(hipFree(dests_dev[i]));
    cudaCheckError(hipFree(block_sums[i]));
  }

  // Compare with reference implementation
  std::unique_ptr<int[]> dest_reference(new int[COUNT]);
  for (int i = 0; i < N_STREAMS; i++) {
    scan_reference(sources[i], dest_reference.get(), COUNT);
    for (int j = 0; j < COUNT; j++) {
      assert(dest_reference.get()[j] == dests[i][j]);
    }
  }

  return 0;
}
